#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>

#include "DataContainer.cuh"
#include "DataFileReader.cuh"
#include "KernelArray.h"

#include <thrust/fill.h>
#include <thrust/remove.h>

DataContainer::DataContainer(HostDataVector& host_data, DeviceDataVector& device_data, unsigned int classes_amount, unsigned int alive_flag_position):
    m_host_data(host_data), m_device_data(device_data), m_classes_amount(classes_amount), m_alive_flag_pos(alive_flag_position),
    m_device_covered_indexes( m_device_data.size() / (m_alive_flag_pos + 1) ) // max possible covery size
{

}

DataContainer::~DataContainer()
{

}

HostDataVector& DataContainer::getHostData() const
{
    return m_host_data;
}

DeviceDataVector& DataContainer::getDeviceData() const
{
    return m_device_data;
}

__global__ void cn2_count_kernel(KernelArray<float>dev_data, KernelArray<Selector> sel_arr, unsigned int flag_position, KernelArray<unsigned int> res_arr,
                                 KernelArray<int> covered_indexes/*, KernelArray<unsigned int> covered_indexes_count*/)
{
     unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

     // | A1 | A2 | ... | An | class_id | flag
     unsigned int thr_flag_pos = (i + 1) * flag_position + i;
     if ( thr_flag_pos < dev_data.m_size && dev_data.m_array[thr_flag_pos] == ALIVE_FLAG ) // check data is alive
     {
         unsigned int j, thr_attr_index;
         int class_index;
         Selector* pSel;
         for (j = 0; j < sel_arr.m_size; ++j) // call all selectors
         {
             pSel = &sel_arr.m_array[j];
             thr_attr_index = pSel->m_attr_index + i * (flag_position + 1);
             if ( !pSel->operator()(dev_data.m_array[ thr_attr_index ]) )
                 break;

         }

         if ( j == sel_arr.m_size ) // all selectors return true
         {

             // increment counter in class position
             class_index = (int) dev_data.m_array[ thr_flag_pos - 1 ];
             atomicAdd(&res_arr.m_array[ class_index ], 1);
             covered_indexes.m_array[i] = thr_flag_pos - flag_position;
         }
     }
}

__global__ void cn2_mark_to_remove_kernel(KernelArray<float>dev_data, KernelArray<Selector> sel_arr, unsigned int flag_position)
{
     unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

     // | A1 | A2 | ... | An | class_id | flag
     unsigned int thr_flag_pos = (i + 1) * flag_position + i;
     if ( thr_flag_pos < dev_data.m_size && dev_data.m_array[thr_flag_pos] == ALIVE_FLAG ) // check data is alive
     {
         unsigned int j, thr_attr_index;

         Selector* pSel;
         for (j = 0; j < sel_arr.m_size; ++j) // call all selectors
         {
             pSel = &sel_arr.m_array[j];
             thr_attr_index = pSel->m_attr_index + i * (flag_position + 1);
             if ( !pSel->operator()(dev_data.m_array[ thr_attr_index ]) )
                 break;
         }

         if ( j == sel_arr.m_size ) // all selectors return true
         {
             // invalidate flag position
             dev_data.m_array[ thr_flag_pos ] = DEAD_FLAG;
         }

     }
}

thrust::host_vector<int> DataContainer::countKernelCall(const thrust::host_vector<Selector>& host_selectors, Distribution& distribution_result_out )
{
    thrust::device_vector<unsigned int> distribution_result(m_classes_amount);
    size_t max_possible_covery_size = m_device_covered_indexes.size();
    thrust::fill(m_device_covered_indexes.begin(), m_device_covered_indexes.end(), -1);
    //thrust::device_vector<int> device_covered_indexes(max_possible_covery_size, -1);

    m_device_selectors = host_selectors;
    cn2_count_kernel<<< ( max_possible_covery_size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>
              (m_device_data, m_device_selectors, m_alive_flag_pos, distribution_result, m_device_covered_indexes);

    hipDeviceSynchronize();
    distribution_result_out = distribution_result;
    //m_device_covered_indexes_end = thrust::remove(m_device_covered_indexes.begin(), m_device_covered_indexes.end(), -1);

    return thrust::host_vector<int>(m_device_covered_indexes.begin(), m_device_covered_indexes.end());
}

void DataContainer::removeKernelCall(const thrust::host_vector<Selector>& host_selectors)
{
    m_device_selectors = host_selectors;
    size_t objects_count = m_device_data.size() / (m_alive_flag_pos + 1);
    cn2_mark_to_remove_kernel<<< ( objects_count + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(m_device_data, m_device_selectors, m_alive_flag_pos);
    hipDeviceSynchronize();
}
