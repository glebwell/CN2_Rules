#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>

#include "DataContainer.cuh"
#include "DataFileReader.cuh"
#include "KernelArray.h"

#include <thrust/fill.h>
#include <thrust/remove.h>

DataContainer::DataContainer(HostDataVector& host_data, DeviceDataVector& device_data, unsigned int classes_amount, unsigned int alive_flag_position):
    m_host_data(host_data), m_device_data(device_data), m_classes_amount(classes_amount), m_alive_flag_pos(alive_flag_position),
    m_device_offsets_buffer( m_device_data.size() / (m_alive_flag_pos + 1) ) // max possible covery size
{

}

DataContainer::~DataContainer()
{

}

HostDataVector& DataContainer::getHostData() const
{
    return m_host_data;
}

DeviceDataVector& DataContainer::getDeviceData() const
{
    return m_device_data;
}

__device__ bool call_selectors(KernelArray<float>& dev_data, KernelArray<Selector>& sel_arr, unsigned int flag_position, unsigned int thread_id, unsigned int thread_flag_pos)
{
    // | A1 | A2 | ... | An | class_id | flag
    if ( thread_flag_pos < dev_data.m_size && dev_data.m_array[thread_flag_pos] == ALIVE_FLAG ) // check data is alive
    {
        unsigned int j, thread_attr_index;
        Selector* pSel;
        for (j = 0; j < sel_arr.m_size; ++j) // call all selectors
        {
            pSel = &sel_arr.m_array[j];
            thread_attr_index = pSel->m_attr_index + thread_id * (flag_position + 1);
            if ( !pSel->operator()(dev_data.m_array[ thread_attr_index ]) )
                break;
        }

        return j == sel_arr.m_size; // all selectors return true
    }
    return false;
}

__global__ void cn2_class_dist_kernel(KernelArray<float>dev_data, KernelArray<Selector> sel_arr, unsigned int flag_position, KernelArray<unsigned int> class_dist_out_res)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int thr_flag_pos = (i + 1) * flag_position + i;
    bool success = call_selectors(dev_data, sel_arr, flag_position, i, thr_flag_pos);
    if ( success )
    {
        int class_index = (int) dev_data.m_array[ thr_flag_pos - 1 ];
        atomicAdd(&class_dist_out_res.m_array[ class_index ], 1);
    }

}

__global__ void cn2_offsets_kernel(KernelArray<float>dev_data, KernelArray<Selector> sel_arr, unsigned int flag_position, KernelArray<int> covered_indexes)
{
     unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
     unsigned int thr_flag_pos = (i + 1) * flag_position + i;
     bool success = call_selectors(dev_data, sel_arr, flag_position, i, thr_flag_pos);
     if ( success )
     {
         covered_indexes.m_array[i] = thr_flag_pos - flag_position;
     }

}

__global__ void cn2_mark_to_remove_kernel(KernelArray<float>dev_data, KernelArray<Selector> sel_arr, unsigned int flag_position)
{
     unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
     unsigned int thr_flag_pos = (i + 1) * flag_position + i;
     bool success = call_selectors(dev_data, sel_arr, flag_position, i, thr_flag_pos);
     if ( success )
     {
         dev_data.m_array[ thr_flag_pos ] = DEAD_FLAG;
     }
}

__global__ void cn2_mark_to_remove_kernel_opt(KernelArray<float>dev_data, KernelArray<int> offsets, unsigned int flag_position)
{
     unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
     if ( i < offsets.m_size )
     {
         int offset = offsets.m_array[ i ];
         if ( offset != -1 )
            dev_data.m_array[ offset + flag_position ] = DEAD_FLAG;
     }
}


Distribution DataContainer::classDistKernelCall(const thrust::host_vector<Selector> &host_selectors)
{
    thrust::device_vector<unsigned int> distribution_result(m_classes_amount);
    m_device_selectors_buffer = host_selectors;
    size_t objects_count = m_device_data.size() / (m_alive_flag_pos + 1);
    cn2_class_dist_kernel<<< ( objects_count + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>
              (m_device_data, m_device_selectors_buffer, m_alive_flag_pos, distribution_result);

    hipDeviceSynchronize();
    Distribution distribution_result_out = distribution_result;

    return distribution_result_out;
}

thrust::host_vector<int> DataContainer::offsetsKernelCall(const thrust::host_vector<Selector> &host_selectors)
{
    size_t max_possible_covery_size = m_device_offsets_buffer.size();
    thrust::fill(m_device_offsets_buffer.begin(), m_device_offsets_buffer.end(), -1);

    m_device_selectors_buffer = host_selectors;
    cn2_offsets_kernel<<< ( max_possible_covery_size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>
              (m_device_data, m_device_selectors_buffer, m_alive_flag_pos, m_device_offsets_buffer);

    hipDeviceSynchronize();
    /*
    auto device_covered_offsets_end = m_device_covered_offsets.end();
    auto device_covered_offsets_new_end = thrust::remove(m_device_covered_offsets.begin(), device_covered_offsets_end, -1);
    device_covered_offsets_new_end = m_device_covered_offsets.erase(device_covered_offsets_new_end, device_covered_offsets_end);
    */
    thrust::host_vector<int> result = m_device_offsets_buffer;
    return result;
}


void DataContainer::removeKernelCall(const thrust::host_vector<Selector> &host_selectors)
{
    m_device_selectors_buffer = host_selectors;
    size_t objects_count = m_device_data.size() / (m_alive_flag_pos + 1);
    cn2_mark_to_remove_kernel<<< ( objects_count + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(m_device_data, m_device_selectors_buffer, m_alive_flag_pos);
    hipDeviceSynchronize();
}


void DataContainer::removeKernelCallOpt(const thrust::host_vector<int> &host_offsets)
{
    m_device_offsets_buffer = host_offsets;
    size_t threads_count = host_offsets.size();
    cn2_mark_to_remove_kernel_opt<<< ( threads_count + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(m_device_data, m_device_offsets_buffer, m_alive_flag_pos);
    hipDeviceSynchronize();
}
