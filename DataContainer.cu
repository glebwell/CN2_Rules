#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>

#include "DataContainer.cuh"
#include "DataFileReader.cuh"
#include "KernelArray.h"

DataContainer::DataContainer(HostDataVector& host_data, DeviceDataVector& device_data, unsigned int classes_amount, unsigned int alive_flag_position):
    m_host_data(host_data), m_device_data(device_data), m_classes_amount(classes_amount), m_alive_flag_pos(alive_flag_position), //m_deleted_objects_count(0),
    m_device_covered_indexes( m_device_data.size() / (m_alive_flag_pos + 1) ) // max possible covery size
{
    //hipMalloc( (void**) &m_device_covered_indexes_count, sizeof(unsigned int) );
}

DataContainer::~DataContainer()
{
    //hipFree(m_device_covered_indexes_count);
}

HostDataVector& DataContainer::getHostData() const
{
    return m_host_data;
}

DeviceDataVector& DataContainer::getDeviceData() const
{
    return m_device_data;
}

__global__ void cn2_count_kernel(KernelArray<float>dev_data, KernelArray<Selector> sel_arr, unsigned int flag_position, KernelArray<unsigned int> res_arr,
                                 KernelArray<unsigned int> covered_indexes, KernelArray<unsigned int> covered_indexes_count)
{
     unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

     // | A1 | A2 | ... | An | class_id | flag
     unsigned int thr_flag_pos = (i + 1) * flag_position + i;
     //if (thr_flag_pos < dev_data_size)
        //printf("{%u, %f}\n", thr_flag_pos, dev_data[thr_flag_pos]);
     if ( thr_flag_pos < dev_data.m_size && dev_data.m_array[thr_flag_pos] == ALIVE_FLAG ) // check data is alive
     {
         //printf("[%u]{%u*%u+%u} ", i, blockIdx.x, blockDim.x, threadIdx.x);
         //printf("[value = %f, index = %d, type = %d]", sel_arr.m_array[0].m_value, (int) sel_arr.m_array[0].m_attr_index, (int) sel_arr.m_array[0].m_type);
         //if (thr_flag_pos == 4)
            //printf("<%u, %f>\n", thr_flag_pos, dev_data[thr_flag_pos]);
         unsigned int j, thr_attr_index;
         int class_index;
         Selector* pSel;
         for (j = 0; j < sel_arr.m_size; ++j) // call all selectors
         {
             pSel = &sel_arr.m_array[j];
             thr_attr_index = pSel->m_attr_index + i * (flag_position + 1);
             //printf("[tfp = <%u>; thr_attr_idx = <%u>; class_idx = <%d>]\n", thr_flag_pos, thr_attr_index, (int) dev_data.m_array[ thr_flag_pos - 1 ]);
             if ( !pSel->operator()(dev_data.m_array[ thr_attr_index ]) )
                 break;

         }
         //printf("[j = %u] [size = %u]", j, sel_arr.m_size - 1);
         if ( j == sel_arr.m_size ) // all selectors return true
         {

             // increment counter in class position
             class_index = (int) dev_data.m_array[ thr_flag_pos - 1 ];
             atomicAdd(&res_arr.m_array[ class_index ], 1);
             covered_indexes.m_array[i] = thr_flag_pos - flag_position;
             atomicAdd(&covered_indexes_count.m_array[0], 1);

             //printf("covered_indexes_count = <%u>", covered_indexes_count.m_array[0]);
         }
     }
}

__global__ void cn2_mark_to_remove_kernel(KernelArray<float>dev_data, KernelArray<Selector> sel_arr, unsigned int flag_position)
{
     unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

     // | A1 | A2 | ... | An | class_id | flag
     unsigned int thr_flag_pos = (i + 1) * flag_position + i;
     //if (thr_flag_pos < dev_data_size)
        //printf("{%u, %f}\n", thr_flag_pos, dev_data[thr_flag_pos]);
     if ( thr_flag_pos < dev_data.m_size && dev_data.m_array[thr_flag_pos] == ALIVE_FLAG ) // check data is alive
     {
         //printf("[%u]{%u*%u+%u} ", i, blockIdx.x, blockDim.x, threadIdx.x);
         //printf("[value = %f, index = %d, type = %d]", sel_arr.m_array[0].m_value, (int) sel_arr.m_array[0].m_attr_index, (int) sel_arr.m_array[0].m_type);
         //if (thr_flag_pos == 4)
            //printf("<%u, %f>\n", thr_flag_pos, dev_data[thr_flag_pos]);
         unsigned int j, thr_attr_index;

         Selector* pSel;
         for (j = 0; j < sel_arr.m_size; ++j) // call all selectors
         {
             pSel = &sel_arr.m_array[j];
             thr_attr_index = pSel->m_attr_index + i * (flag_position + 1);
             //printf("[%u %u %u %d]\n", dev_data_size, thr_flag_pos, thr_attr_index, (int) dev_data[ thr_flag_pos - 1 ]);
             if ( !pSel->operator()(dev_data.m_array[ thr_attr_index ]) )
                 break;

         }
         //printf("[j = %u] [size = %u]", j, sel_arr.m_size - 1);
         if ( j == sel_arr.m_size ) // all selectors return true
         {
             // invalidate flag position
             dev_data.m_array[ thr_flag_pos ] = DEAD_FLAG;
         }

     }
}

HostDataVector DataContainer::countKernelCall(const thrust::host_vector<Selector>& host_selectors, Distribution& distribution_result_out )
{
    thrust::device_vector<unsigned int> distribution_result(m_classes_amount);
    size_t max_possible_covery_size = m_device_covered_indexes.size();
    thrust::device_vector<unsigned int> dev_counter(1);
    m_device_selectors = host_selectors;
    cn2_count_kernel<<< ( max_possible_covery_size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>
              (m_device_data, m_device_selectors, m_alive_flag_pos, distribution_result, m_device_covered_indexes, dev_counter);
    hipDeviceSynchronize();
    distribution_result_out = distribution_result;
    thrust::host_vector<unsigned int> host_counter = dev_counter;
    return HostDataVector(m_device_covered_indexes.begin(), m_device_covered_indexes.begin() + host_counter[0]);
}

void DataContainer::removeKernelCall(const thrust::host_vector<Selector>& host_selectors)
{
    m_device_selectors = host_selectors;
    size_t objects_count = m_device_data.size() / (m_alive_flag_pos + 1);
    cn2_mark_to_remove_kernel<<< ( objects_count + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(m_device_data, m_device_selectors, m_alive_flag_pos);
    hipDeviceSynchronize();
}
