#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>

#include "DataContainer.cuh"
#include "DataFileReader.cuh"
#include "KernelArray.h"

#include <thrust/fill.h>
#include <thrust/remove.h>

DataContainer::DataContainer(HostDataVector& host_data, DeviceDataVector& device_data, unsigned int classes_amount, unsigned int alive_flag_position):
    m_host_data(host_data), m_device_data(device_data), m_classes_amount(classes_amount), m_alive_flag_pos(alive_flag_position), //m_deleted_objects_count(0),
    m_device_covered_indexes( m_device_data.size() / (m_alive_flag_pos + 1) ) // max possible covery size
{
    //hipMalloc( (void**) &m_device_covered_indexes_count, sizeof(unsigned int) );
}

DataContainer::~DataContainer()
{
    //hipFree(m_device_covered_indexes_count);
}

HostDataVector& DataContainer::getHostData() const
{
    return m_host_data;
}

DeviceDataVector& DataContainer::getDeviceData() const
{
    return m_device_data;
}

__global__ void cn2_count_kernel(KernelArray<float>dev_data, KernelArray<Selector> sel_arr, unsigned int flag_position, KernelArray<unsigned int> res_arr,
                                 KernelArray<int> covered_indexes/*, KernelArray<unsigned int> covered_indexes_count*/)
{
     unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

     // | A1 | A2 | ... | An | class_id | flag
     unsigned int thr_flag_pos = (i + 1) * flag_position + i;
     //if (thr_flag_pos < dev_data_size)
        //printf("{%u, %f}\n", thr_flag_pos, dev_data[thr_flag_pos]);
     if ( thr_flag_pos < dev_data.m_size && dev_data.m_array[thr_flag_pos] == ALIVE_FLAG ) // check data is alive
     {
         //printf("[%u]{%u*%u+%u} ", i, blockIdx.x, blockDim.x, threadIdx.x);
         //printf("[value = %f, index = %d, type = %d]", sel_arr.m_array[0].m_value, (int) sel_arr.m_array[0].m_attr_index, (int) sel_arr.m_array[0].m_type);
         //if (thr_flag_pos == 4)
            //printf("<%u, %f>\n", thr_flag_pos, dev_data[thr_flag_pos]);
         unsigned int j, thr_attr_index;
         int class_index;
         Selector* pSel;
         for (j = 0; j < sel_arr.m_size; ++j) // call all selectors
         {
             pSel = &sel_arr.m_array[j];
             thr_attr_index = pSel->m_attr_index + i * (flag_position + 1);
             //printf("[tfp = <%u>; thr_attr_idx = <%u>; class_idx = <%d>]\n", thr_flag_pos, thr_attr_index, (int) dev_data.m_array[ thr_flag_pos - 1 ]);
             if ( !pSel->operator()(dev_data.m_array[ thr_attr_index ]) )
                 break;

         }
         //printf("[j = %u] [size = %u]", j, sel_arr.m_size - 1);
         if ( j == sel_arr.m_size ) // all selectors return true
         {

             // increment counter in class position
             class_index = (int) dev_data.m_array[ thr_flag_pos - 1 ];
             atomicAdd(&res_arr.m_array[ class_index ], 1);
             covered_indexes.m_array[i] = thr_flag_pos - flag_position;
             //atomicAdd(&covered_indexes_count.m_array[0], 1);

             //printf("i = <%u> thr_flag_pos = <%u> inserted <%u>", i, thr_flag_pos, thr_flag_pos - flag_position);
         }
     }
}

__global__ void cn2_mark_to_remove_kernel(KernelArray<float>dev_data, KernelArray<Selector> sel_arr, unsigned int flag_position)
{
     unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

     // | A1 | A2 | ... | An | class_id | flag
     unsigned int thr_flag_pos = (i + 1) * flag_position + i;
     //if (thr_flag_pos < dev_data_size)
        //printf("{%u, %f}\n", thr_flag_pos, dev_data[thr_flag_pos]);
     if ( thr_flag_pos < dev_data.m_size && dev_data.m_array[thr_flag_pos] == ALIVE_FLAG ) // check data is alive
     {
         //printf("[%u]{%u*%u+%u} ", i, blockIdx.x, blockDim.x, threadIdx.x);
         //printf("[value = %f, index = %d, type = %d]", sel_arr.m_array[0].m_value, (int) sel_arr.m_array[0].m_attr_index, (int) sel_arr.m_array[0].m_type);
         //if (thr_flag_pos == 4)
            //printf("<%u, %f>\n", thr_flag_pos, dev_data[thr_flag_pos]);
         unsigned int j, thr_attr_index;

         Selector* pSel;
         for (j = 0; j < sel_arr.m_size; ++j) // call all selectors
         {
             pSel = &sel_arr.m_array[j];
             thr_attr_index = pSel->m_attr_index + i * (flag_position + 1);
             //printf("[%u %u %u %d]\n", dev_data_size, thr_flag_pos, thr_attr_index, (int) dev_data[ thr_flag_pos - 1 ]);
             if ( !pSel->operator()(dev_data.m_array[ thr_attr_index ]) )
                 break;

         }
         //printf("[j = %u] [size = %u]", j, sel_arr.m_size - 1);
         if ( j == sel_arr.m_size ) // all selectors return true
         {
             // invalidate flag position
             dev_data.m_array[ thr_flag_pos ] = DEAD_FLAG;
         }

     }
}

thrust::host_vector<int> DataContainer::countKernelCall(const thrust::host_vector<Selector>& host_selectors, Distribution& distribution_result_out )
{
    thrust::device_vector<unsigned int> distribution_result(m_classes_amount);
    size_t max_possible_covery_size = m_device_covered_indexes.size();
    //thrust::fill(m_device_covered_indexes.begin(), m_device_covered_indexes.end(), -1);
    thrust::device_vector<int> device_covered_indexes(max_possible_covery_size, -1);
    //thrust::device_vector<unsigned int> dev_counter(1);
    m_device_selectors = host_selectors;
    cn2_count_kernel<<< ( max_possible_covery_size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>
              (m_device_data, m_device_selectors, m_alive_flag_pos, distribution_result, device_covered_indexes/*, dev_counter*/);
    hipDeviceSynchronize();
    distribution_result_out = distribution_result;
    //thrust::host_vector<unsigned int> host_counter = dev_counter;
    //auto new_end = thrust::remove(device_covered_indexes.begin(), device_covered_indexes.end(), -1
                                     //__device__ [] (int v) {return v == -1;}
    //);

    //thrust::host_vector<int> result(device_covered_indexes.begin(), device_covered_indexes.end());
    //std::vector<int> vec_offsets(result.begin(), result.end());
    //std::vector<unsigned int> vec_dist(distribution_result_out.begin(), distribution_result_out.end());
    return thrust::host_vector<int>(device_covered_indexes.begin(), device_covered_indexes.end());
}

void DataContainer::removeKernelCall(const thrust::host_vector<Selector>& host_selectors)
{
    m_device_selectors = host_selectors;
    //std::vector<Selector> selectors_check(host_selectors.cbegin(), host_selectors.cend());
    size_t objects_count = m_device_data.size() / (m_alive_flag_pos + 1);
    cn2_mark_to_remove_kernel<<< ( objects_count + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(m_device_data, m_device_selectors, m_alive_flag_pos);
    //std::vector<float> data_check(m_device_data.cbegin(), m_device_data.cend());
    hipDeviceSynchronize();
}
