#include <iostream>
#include "DataFileReader.cuh"
#include "CN2UnorderedLearner.h"
#include "RuleClassifier.h"
#include "Measure.h"


//CN2UnorderedLearner learner;

void train()
{
    //learner.fit(DataFileReader::getInstance().trainData());
    //learner.printRules();
}
void test()
{
    //RuleClassifier rc(DataFileReader::getInstance().testData(), learner.rules());
    //rc.run();
}
int main(int argc, char* argv[])
{
    const char* filename = argv[1];
	try
	{
        DataFileReader::run(filename, 3000000);
        //std::cout << measure<>::execution(train) <<"ms" << "\n";
        //std::cout << measure<>::execution(test) << "ms" << "\n";
        thrust::host_vector<float>& h_vec = DataFileReader::getInstance().trainData();
        size_t line = DataFileReader::getInstance().attributes().size() + 2;
        for (size_t i = 0; i < h_vec.size(); ++i)
        {
            if (i != 0 && (i % line) == 0 )
                std::cout << "\n";
            std::cout  << h_vec[i] << " ";
        }

        std::cout << "\n";
        DataFileReader::freeDeviceData();
	}
	catch (const std::exception& e)
	{
		std::cout << e.what() << "\n";
	}

	
	return 0;
}
