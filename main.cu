#include <iostream>
#include "DataFileReader.cuh"
#include "CN2UnorderedLearner.h"
#include "Measure.h"
#include "DataContainer.cuh"

int main(int argc, char* argv[])
{

    if ( argc != 5 )
    {
        std::cout << "usage: <program> <filename> <max_rule_length> <beam_width> <min_rule_conf>\n";
    }
    else
    {
        try
        {
            const char* filename = argv[1];
            unsigned int max_rule_length = std::stoul( argv[2] );
            unsigned char beam_width = std::stoul( argv[3] );
            float min_rule_conf = std::stof( argv[4] );

            DataFileReader::run(filename, -1);
            DataFileReader& fr = DataFileReader::getInstance();
            GuardianValidator::setMaxRuleLength(max_rule_length);
            unsigned int alive_flag_position = fr.attributes().size() + 1; // attributes + class + flag
            DataContainer data(fr.trainData(), fr.deviceData(), fr.distribution().size(), alive_flag_position);
            CN2UnorderedLearner learner(min_rule_conf, beam_width);
            learner.fit(data);
            learner.printRules();

            std::cout << "Total rules : " << learner.rulesCount() << "\n";
            std::cout << "Average quality : " << learner.averageQuality() << "\n";
            std::cout << "Max quality : " << learner.maxQuality() << "\n";
            std::cout << "Database coverage: " << learner.databaseCoverage() * 100 << "%\n";

            DataFileReader::freeDeviceData();
        }
        catch (const std::exception& e)
        {
            std::cout << e.what() << "\n";
        }
    }

	return 0;
}
