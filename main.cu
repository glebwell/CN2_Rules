#include <iostream>
#include "DataFileReader.cuh"
#include "CN2UnorderedLearner.h"
//#include "RuleClassifier.h"
#include "Measure.h"
#include "DataContainer.cuh"

//CN2UnorderedLearner learner;

void train()
{
    //learner.fit(DataFileReader::getInstance().trainData());
    //learner.printRules();
}
void test()
{
    //RuleClassifier rc(DataFileReader::getInstance().testData(), learner.rules());
    //rc.run();
}
int main(int argc, char* argv[])
{
    const char* filename = argv[1];
    if ( filename )
    {
        try
        {
            DataFileReader::run(filename, 3000000);
            DataFileReader& fr = DataFileReader::getInstance();
            unsigned int alive_flag_position = fr.attributes().size() + 2; // attributes + class + flag
            DataContainer data(fr.trainData(), fr.deviceData(), fr.distribution().size(), alive_flag_position);
            CN2UnorderedLearner learner;
            learner.fit(data);
            learner.printRules();

            //std::cout << measure<>::execution(train) <<"ms" << "\n";
            //std::cout << measure<>::execution(test) << "ms" << "\n";
            /*
            thrust::host_vector<float>& h_vec = DataFileReader::getInstance().trainData();
            size_t line = DataFileReader::getInstance().attributes().size() + 2;
            for (size_t i = 0; i < h_vec.size(); ++i)
            {
                if (i != 0 && (i % line) == 0 )
                    std::cout << "\n";
                std::cout  << h_vec[i] << " ";
            }

            std::cout << "\n";
            */
            DataFileReader::freeDeviceData();
        }
        catch (const std::exception& e)
        {
            std::cout << e.what() << "\n";
        }
    }
    else
    {
        std::cout << "input file is not set" << std::endl;
    }

	
	return 0;
}
